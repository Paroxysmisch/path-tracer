#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include <cstdlib>
#include <new>
#include <hiprand/hiprand_kernel.h>
#include "brdf.cuh"
#include "check_cuda_errors.h"
#include "constants.h"
#include "util.cuh"
#include "world.cuh"
#include "shapes.cuh"
#include "phong.cuh"
#include "camera.cuh"
#include "denoise.cuh"

// __device__ pathtracer::vec3 trace_path(pathtracer::ray ray, int depth, pathtracer::world world, int* collision_buffer, pathtracer::intersection* intersection_buffer, hiprandState* state) {
//     if (depth <= 0) return {0.f, 0.f, 0.f};

//     bool success_flag{false};

//     pathtracer::computations comp = world.intersect_world(ray, success_flag, collision_buffer, intersection_buffer);

//     if (!success_flag) return {0.f, 0.f, 0.f};

//     pathtracer::object& object = world.objects[comp.intersection.object_index];

//     if (object.mat_t == pathtracer::LIGHT) {
//         return {1.f, 1.f, 1.f};
//     }

//     float u = hiprand_uniform(state);
//     float v = hiprand_uniform(state);
//     float pdf;

//     pathtracer::point new_direction = pathtracer::cosine_sample_hemisphere(u, v, pdf);

//     pathtracer::quaternion q_to_world = pathtracer::quaternion::get_rotation_from_z_axis(comp.surface_normal.normalize());

//     new_direction = pathtracer::quaternion::rotate_vector_by_quaternion(new_direction, q_to_world) + pathtracer::vec3(0.01f, 0.01f, 0.01f);

//     pathtracer::ray new_ray{comp.surface_point, new_direction.normalize()};

//     float cos_theta = new_ray.d * comp.surface_normal;
//     pathtracer::vec3 BRDF = object.mat_d.phong.color * object.mat_d.phong.diffuse / pathtracer::pi;

//     pathtracer::vec3 incoming = trace_path(new_ray, depth - 1, world, collision_buffer, intersection_buffer, state);

//     return (BRDF & incoming) * (cos_theta / pdf);
// }

__global__ void constant_brdf_test(pathtracer::canvas c, pathtracer::world world, pathtracer::camera camera, hiprandState* d_states) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    const int j_original = j;
    int num_threads_i = blockDim.y * gridDim.y;
    int num_threads_j = blockDim.x * gridDim.x;

    hiprandState* state = &d_states[i * num_threads_j + j];
    hiprand_init(1234, i * num_threads_j + j, 0, state);

    pathtracer::vec3 color_black{0.f, 0.f, 0.f};

    int collision_buffer_offset = world.num_objects * (i * num_threads_j + j);
    int* collision_buffer = (world.collision_buffer + collision_buffer_offset);

    int intersection_buffer_offset = 2 * world.num_objects * (i * num_threads_j + j);

    pathtracer::intersection* intersection_buffer = (world.intersection_buffer + intersection_buffer_offset);

    constexpr int max_depth = 10;
    constexpr int num_samples = 1000;

    while (i < 1000) {
        while (j < 1000) {
            pathtracer::vec3 color{0.f, 0.f, 0.f};

            for (int k{0}; k < num_samples; ++k) {
                // floats a and b for anti-aliasing
                float a = hiprand_uniform(state);
                float b = hiprand_uniform(state);
                pathtracer::ray ray = camera.gen_ray_for_pixel(i, j, a, b);
                bool success_flag{false};
                pathtracer::vec3 multiplier{1.f, 1.f, 1.f};
                multiplier *= pathtracer::one_over_pi;

                for (int l{0}; l < max_depth; ++l) {
                    pathtracer::computations comp = world.intersect_world(ray, success_flag, collision_buffer, intersection_buffer);

                    if (!success_flag) {
                        multiplier &= {0.f, 0.f, 0.f};
                        break;
                    }

                    pathtracer::object& object = world.objects[comp.intersection.object_index];

                    float u = hiprand_uniform(state);
                    float v = hiprand_uniform(state);
                    // float pdf;

                    // pathtracer::point new_direction = pathtracer::cosine_sample_hemisphere(u, v, pdf);

                    // pathtracer::quaternion q_to_world = pathtracer::quaternion::get_rotation_from_z_axis(comp.surface_normal.normalize());

                    // new_direction = pathtracer::quaternion::rotate_vector_by_quaternion(new_direction, q_to_world);

                    // ray = pathtracer::ray(comp.surface_point + (comp.surface_normal * 0.01f), new_direction.normalize());

                    // float cos_theta = ray.d * comp.surface_normal;

                    if (object.mat_t == pathtracer::LIGHT) {
                        multiplier &= {500.f, 500.f, 500.f};
                        break;
                    }

                    // color += multiplier & (object.mat_d.phong.color * object.mat_d.phong.ambient) * cos_theta;

                    // multiplier &= (object.mat_d.phong.color * object.mat_d.phong.diffuse) * (cos_theta / pdf) * pathtracer::one_over_pi;

                    pathtracer::vector out_ray_direction;

                    pathtracer::vector out_sample_weight;

                    bool eval_successful = pathtracer::eval_brdf(u, v, comp.surface_normal, comp.eye_vector, out_ray_direction, out_sample_weight, object.mat_d.microfacet);

                    if (!eval_successful) break;

                    multiplier &= out_sample_weight;

                    ray = pathtracer::ray(comp.surface_point + (comp.surface_normal * 0.01f), out_ray_direction);
                }

                color += multiplier;
            }

            color /= num_samples;
                
            c.write_pixel(i, j, color);
            
            j += num_threads_j;
        }
        i += num_threads_i;
        j = j_original;
    }
}

TEST_CASE("Full brdf renders") {
    SECTION("Constant") {
        constexpr int canvas_pixels = 1000;
        pathtracer::canvas c{canvas_pixels, canvas_pixels};

        dim3 blocks(16, 16);
        dim3 threads(16, 16);

        pathtracer::camera camera(1000, 1000, pathtracer::pi / 2.f, {0.f, 0.f, -10.f}, {0.f, 0.f, 0.f}, {0.f, 1.f, 0.f}, pathtracer::mat4::get_rotation_z(pathtracer::pi / 4.f));

        // pathtracer::world w({
        //     {pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(-2.f, 0.f, -2.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.25f, 0.25f, 0.95f}, 0.1f, 0.9f, 0.9f, 200)},
        //     {pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(-1.f, -1.f, 0.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.35f, 0.25f, 0.75f}, 0.1f, 0.9f, 0.9f, 200)},
        //     {pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(0.f, 0.f, -1.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.75f, 0.25f, 0.5f}, 0.1f, 0.9f, 0.9f, 100)},
        //     {pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(1.f, 1.f, 2.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.75f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)},
        //     {pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(2.f, 0.f, 1.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.95f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)},
        //     {pathtracer::SPHERE,
        //      pathtracer::sphere(pathtracer::mat4::get_translation(-10.f, 0.f, -10.f)),
        //      pathtracer::LIGHT,
        //      pathtracer::phong({0.95f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)},
        //     {pathtracer::SPHERE,
        //      pathtracer::sphere(pathtracer::mat4::get_translation(10.f, 0.f, -10.f)),
        //      pathtracer::LIGHT,
        //      pathtracer::phong({0.95f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)}
        // }, blocks, threads);

        // pathtracer::object obj0{pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(-2.f, 0.f, -2.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.25f, 0.25f, 0.95f}, 0.1f, 0.9f, 0.9f, 200)};
        // pathtracer::object obj1{pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(-1.f, -1.f, 0.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.35f, 0.25f, 0.75f}, 0.1f, 0.9f, 0.9f, 200)};
        // pathtracer::object obj2{pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(0.f, 0.f, -1.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.75f, 0.25f, 0.5f}, 0.1f, 0.9f, 0.9f, 100)};
        // pathtracer::object obj3{pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(1.f, 1.f, 2.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.75f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)};
        // pathtracer::object obj4{pathtracer::SPHERE, 
        //      pathtracer::sphere(pathtracer::mat4::get_translation(2.f, 0.f, 1.f)),
        //      pathtracer::PHONG,
        //      pathtracer::phong({0.95f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)};
        // pathtracer::object obj5{pathtracer::SPHERE,
        //      pathtracer::sphere(pathtracer::mat4::get_translation(-10.f, 0.f, -10.f)),
        //      pathtracer::LIGHT,
        //      pathtracer::phong({0.95f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)};
        // pathtracer::object obj6{pathtracer::SPHERE,
        //      pathtracer::sphere(pathtracer::mat4::get_translation(10.f, 0.f, -10.f)),
        //      pathtracer::LIGHT,
        //      pathtracer::phong({0.95f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)};

        pathtracer::object obj0{pathtracer::SPHERE, 
             pathtracer::sphere(pathtracer::mat4::get_translation(-2.f, 0.f, -2.f)),
             pathtracer::MICROFACET,
             pathtracer::phong{{0.f, 0.f, 0.f}, 0.f, 0.f, 0.f, 0.f}};
        obj0.mat_d.microfacet = pathtracer::microfacet{{0.25f, 0.25f, 0.95f}, {0.f, 0.f, 0.f}, 0.75f, 0.2f, 0.f};

        pathtracer::object obj1{pathtracer::SPHERE, 
             pathtracer::sphere(pathtracer::mat4::get_translation(-1.f, -1.f, 0.f)),
             pathtracer::MICROFACET,
             pathtracer::phong{{0.f, 0.f, 0.f}, 0.f, 0.f, 0.f, 0.f}};
        obj1.mat_d.microfacet = pathtracer::microfacet{{0.35f, 0.25f, 0.75f}, {0.f, 0.f, 0.f}, 0.75f, 0.2f, 0.f};
        
        pathtracer::object obj2{pathtracer::SPHERE, 
             pathtracer::sphere(pathtracer::mat4::get_translation(0.f, 0.f, -1.f)),
             pathtracer::MICROFACET,
             pathtracer::phong{{0.f, 0.f, 0.f}, 0.f, 0.f, 0.f, 0.f}};
        obj2.mat_d.microfacet = pathtracer::microfacet{{0.75f, 0.25f, 0.5f}, {0.f, 0.f, 0.f}, 0.75f, 0.2f, 0.f};

        pathtracer::object obj3{pathtracer::SPHERE, 
             pathtracer::sphere(pathtracer::mat4::get_translation(1.f, 1.f, 2.f)),
             pathtracer::MICROFACET,
             pathtracer::phong{{0.f, 0.f, 0.f}, 0.f, 0.f, 0.f, 0.f}};
        obj3.mat_d.microfacet = pathtracer::microfacet{{0.75f, 0.25f, 0.5f}, {0.f, 0.f, 0.f}, 0.75f, 0.2f, 0.f};

        pathtracer::object obj4{pathtracer::SPHERE, 
             pathtracer::sphere(pathtracer::mat4::get_translation(2.f, 0.f, 1.f)),
             pathtracer::MICROFACET,
             pathtracer::phong{{0.f, 0.f, 0.f}, 0.f, 0.f, 0.f, 0.f}};
        obj4.mat_d.microfacet = pathtracer::microfacet{{0.95f, 0.25f, 0.5f}, {0.f, 0.f, 0.f}, 0.75f, 0.2f, 0.f};

        pathtracer::object obj5{pathtracer::SPHERE,
             pathtracer::sphere(pathtracer::mat4::get_translation(-10.f, 0.f, -10.f)),
             pathtracer::LIGHT,
             pathtracer::phong({0.95f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)};

        pathtracer::object obj6{pathtracer::SPHERE,
             pathtracer::sphere(pathtracer::mat4::get_translation(10.f, 0.f, -10.f)),
             pathtracer::LIGHT,
             pathtracer::phong({0.95f, 0.25f, 0.5f}, 0.3, 0.7, 0.5, 10)};


        pathtracer::world w({
            &obj0, &obj1, &obj2, &obj3, &obj4, &obj5, &obj6
        }, blocks, threads);

        hiprandState* d_states;

        checkCudaErrors( hipMalloc(reinterpret_cast<void**>(&d_states), blocks.y * blocks.x * threads.y * threads.x * sizeof(hiprandState)) );

        constant_brdf_test<<<blocks, threads>>>(c, w, camera, d_states);

        checkCudaErrors( hipDeviceSynchronize() );

        c.export_as_PPM("Constant_BRDF_Test_GPU.ppm");
        c.export_as_EXR("Constant_BRDF_Test_GPU.exr");
        pathtracer::denoise(canvas_pixels, canvas_pixels, "Constant_BRDF_Test_GPU.exr", w, camera, "Constant_BRDF_Test_GPU_denoised.exr");
    }
}